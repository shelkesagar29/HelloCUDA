#include "hip/hip_runtime.h"
#include "src/GEMM/GemmNaive.cuh"
#include "src/Utils/Utils.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>

__global__ void gemmNaive(int M, int N, int K, float alpha, const float *A,
                          const float *B, float beta, float *C) {

  const int row = blockIdx.y * blockDim.y + threadIdx.y;
  const int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[row * K + i] * B[i * N + col];
    }
    // C = α*(A@B)+β*C
    C[row * N + col] = alpha * tmp + beta * C[row * N + col];
  }
}

int main() {
  int64_t m{3};
  int64_t k{3};
  int64_t n{3};

  float *A = nullptr, *B = nullptr, *C = nullptr;    // host matrices
  float *dA = nullptr, *dB = nullptr, *dC = nullptr; // device matrices

  A = (float *)malloc(sizeof(float) * m * k);
  B = (float *)malloc(sizeof(float) * k * n);
  C = (float *)malloc(sizeof(float) * m * n);

  fillLinspace(A, 1.0f, m * k, 1.0f);
  fillLinspace(B, 1.0f, k * n, 1.0f);
  fillConstant(C, m * n, 0);
  printMatrix(A, m, k);
  printMatrix(B, k, n);

  CHECK_CUDA_ERROR(hipMalloc((void **)&dA, sizeof(float) * m * k));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dB, sizeof(float) * k * n));
  CHECK_CUDA_ERROR(hipMalloc((void **)&dC, sizeof(float) * m * n));

  CHECK_CUDA_ERROR(
      hipMemcpy(dA, A, sizeof(float) * m * k, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dB, B, sizeof(float) * k * n, hipMemcpyHostToDevice));
  CHECK_CUDA_ERROR(
      hipMemcpy(dC, C, sizeof(float) * m * n, hipMemcpyHostToDevice));

  dim3 blockDim(32, 32);
  dim3 gridDim(CEIL_DIV(m, 32), CEIL_DIV(n, 32));
  float alpha{1.0f};
  float beta{1.0f};

  gemmNaive<<<gridDim, blockDim>>>(m, n, k, alpha, dA, dB, beta, dC);
  CHECK_CUDA_ERROR(hipDeviceSynchronize());
  CHECK_CUDA_ERROR(
      hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost));

  printMatrix(C, m, n);
  free(A);
  free(B);
  free(C);

  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
}