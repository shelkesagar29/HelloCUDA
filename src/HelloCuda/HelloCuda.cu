
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

__global__ void cuda_hello() { printf("Hello CUDA!\n"); }

int main() {

  cuda_hello<<<1, 1>>>();
  hipDeviceSynchronize();
  return 0;
}