
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hip/hip_runtime_api.h>

int main() {
  int deviceId;

  hipGetDevice(&deviceId);

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  printf("Device ID: %d\n\
       *Number of SMs: %d\n\
       Compute Capability Major: %d\n\
       Compute Capability Minor: %d\n\
       memoryBusWidth: %d\n\
       *maxThreadsPerBlock: %d\n\
       maxThreadsPerMultiProcessor: %d\n\
       *totalGlobalMem: %zuM\n\
       sharedMemPerBlock: %zuKB\n\
       *sharedMemPerMultiprocessor: %zuKB\n\
       totalConstMem: %zuKB\n\
       *multiProcessorCount: %d\n\
       *Warp Size: %d\n",
         deviceId, props.multiProcessorCount, props.major, props.minor,
         props.memoryBusWidth, props.maxThreadsPerBlock,
         props.maxThreadsPerMultiProcessor, props.totalGlobalMem / 1024 / 1024,
         props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
  return 0;
};